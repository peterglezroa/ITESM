#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define TPB 10

__global__ void matrix_sub(int size, int *map, int *change) {
    /* Function that updates the map. Each thread will substract the changes to the actual map */
    int index = threadIdx.x + blockIdx.x*blockDim.x; // linearize matrix
    if(index < size) {
        map[index] -= change[index];
        if(map[index] < 0)
            map[index] = 0;
    }
}

void fill_random(int size, int *array) {
    /* Function to make random values on an array from 0 to 10 */
    for(int index = 0; index < size; index++)
        array[index] = rand()%11;
}

__global__ void print_m(int dimx, int dimy, int *m) {
    for(int i = 0; i < dimy; i++) {
        for(int j = 0; j < dimx; j++){
            printf("%i ", m[j + i*dimx]);
        }
        printf("\n");
    }
    printf("\n");
}

void print_m_host(int dimx, int dimy, int *m) {
    for(int i = 0; i < dimy; i++) {
        for(int j = 0; j < dimx; j++){
            printf("%i ", m[j + i*dimx]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
    int *map, *change, size, dimx, dimy, rounds;
    int *d_map, *d_change;

    srand(time(NULL)); // Make random seed

    printf("Give me the x dimension of the map: ");
    scanf("%i", &dimx);

    printf("Give me the y dimension of the map: ");
    scanf("%i", &dimy);

    size = dimx*dimy; // Matrix will be linearized, no need to specify dimensions

    // Initialize HOST pointers
    map = (int *)malloc(sizeof(int)*size);
    fill_random(size, map);
    print_m_host(dimx, dimy, map);
    change = (int *)malloc(sizeof(int)*size);

    // Initialize DEVICE pointers
    hipMalloc((void**)&d_map, sizeof(float)*size);
    hipMemcpy(d_map, map, sizeof(int)*size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_change, sizeof(float)*size);

    // Will not scan and the damage will be random to save time with scans
    printf("How many rounds will be played? ");
    scanf("%i", &rounds);

    printf("Starting map:\n");
    print_m<<<1,1>>>(dimx, dimy, d_map);

    for(int i = 0; i < rounds; i++) {
        // Init map damages for the round and copy to device
        fill_random(size, change);
        hipMemcpy(d_change, change, sizeof(int)*size, hipMemcpyHostToDevice);
        printf("Damage to map at round %i:\n", i);
        print_m_host(dimx, dimy, change);

        // Damage map in device
        matrix_sub<<<(TPB+size)/TPB, TPB>>>(size, d_map, d_change);

        // Copy and print map after round damage
        printf("Map after round %i:\n", i);
        hipMemcpy(map, d_map, sizeof(int)*size, hipMemcpyDeviceToHost);
        print_m_host(dimx, dimy, map);
    }

    free(map); free(change);
    hipFree(d_map); hipFree(d_change);
    return 0;
}
