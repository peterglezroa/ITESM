#include "hip/hip_runtime.h"
/*
   Pedro Luis González Roa
   A01651517
   Cuda Lab 2
*/
// %%cu
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 10

__device__ void matrix_dot(int col, int row, int m, int n, int k, float *d_ma, float *d_mb, float *d_mc) {
    int index = col + row*n;
    d_mc[index] = 0;
    for(int i = 0; i < n; i++) {
        d_mc[index] += d_ma[i + row*n]*d_mb[col+i*n];
    }
}

__global__ void valid_matrix_dot(int m, int n, int k, float *d_ma, float *d_mb, float *d_mc) {
    int col = threadIdx.x + blockIdx.x*blockDim.x;
    int row = threadIdx.y + blockIdx.y*blockDim.y;
    if(m > col && n > row) {
        matrix_dot(col, row, m, n, k, d_ma, d_mb, d_mc);
    }
}

int main() {
    float *ma, *mb, *mc;
    float *d_ma, *d_mb, *d_mc;
    int m, n, n2, k, blocks;

    /* input first matrix */
    fprintf(stdout, "Give me the dimensions of the first matrix: ");
    fscanf(stdin, "%i %i", &m, &n);
    ma = (float *)malloc(sizeof(float)*m*n);

    fprintf(stdout, "Give me the first matrix, separating columns with space and rows with endline.\n");
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            fscanf(stdin, "%f", &ma[j+i*n]);
        }
    }

    /* Input second matrix */
    fprintf(stdout, "Give me the dimensions of the second matrix: ");
    fscanf(stdin, "%i %i", &n2, &k);
    if(n != n2) {
        fprintf(stdout, "Incorrect Dimensions!\n");
        return -1;
    }
    mb = (float *)malloc(sizeof(float)*n*k);

    fprintf(stdout, "Give me the second matrix, separating columns with space and rows with endline.\n");
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < k; j++) {
            fscanf(stdin, "%f", &mb[k+i*n]);
        }
    }

    /* Make space for third matrix */
    mc = (float *)malloc(sizeof(float)*m*k);

    HANDLE_ERROR(hipMalloc((void**)&d_ma, sizeof(float)*m*n));
    HANDLE_ERROR(hipMalloc((void**)&d_mb, sizeof(float)*n*k));
    HANDLE_ERROR(hipMalloc((void**)&d_mc, sizeof(float)*m*k));
    HANDLE_ERROR(hipMemcpy(d_ma, ma, sizeof(float)*m*n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_mb, mb, sizeof(float)*n*k, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_mc, mc, sizeof(float)*m*k, hipMemcpyHostToDevice));

    /* call function */
    blocks = ceil((THREADS_PER_BLOCK+m*k)/THREADS_PER_BLOCK)
    dim3 Blocks(blocks, blocks);
    dim3 Threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    valid_matrix_dot<<<Blocks, Threads>>>(m, n, k, d_ma, d_mb, d_mc);

    /* cpy result */
    hipMemcpy(mc, d_mc, sizeof(float)*m*k, hipMemcpyDeviceToHost);

    /* print result */
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < k; j++) {
            fprintf(stdout, "%f ", mc[i][j]);
        }
        fprintf(stdout, "\n");
    }

    free(ma);free(mb);free(mc);
    hipFree(d_ma);hipFree(d_mb);hipFree(d_mc);
    return 0;
}
