#include "hip/hip_runtime.h"
#include <cuda_runtime>
#include <stdio.h>
#include <time>

#define TPB 10

__global__ void round_change(int size, int *map, int *change) {
    /* Function that updates the map. Each thread will sum the changes to the actual map */
    int index = threadIdx.x + blockIdx.x*blockDim.x; // linearize matrix
    if(index < size)
        map[index] += change[index];
}

__global__ void fill_random(int size, int *array) {
    /* Function to make random values on an array from 0 to 10 */
    int index = threadIdx.x + blockIdx.x*blockDim.x; // linearize matrix
    if(index < size)
        array[index] = rand()%11;
}

int main() {
    int *map, *change, size, dimx, dimy;
    int *d_map, *d_change;

    srand(time(NULL)); // Make random seed

    printf("Give me the x dimension of the map: ");
    scanf("%i", &dimx);

    printf("Give me the y dimension of the map: ");
    scanf("%i", &dimy);

    size = dimx*dimy; // Matrix will be linearized, no need to specify dimensions

    // Initialize HOST pointers
    map = (int *)malloc(sizeof(int)*size);
    change = (int *)malloc(sizeof(int)*size);

    // Initialize DEVICE pointers

    fill_random<<<(TPB+size)/TPB, TPB>>>(size, d_map);
