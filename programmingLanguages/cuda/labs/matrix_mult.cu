/*
   Pedro Luis González Roa
   A01651517
   Cuda Lab 2
*/
// %%cu
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 10

__device__ void matrix_dot(int col, int row, int m, int n, int k, float *d_ma, float *d_mb, float *d_mc) {
    int index = col + row*k; // linearize the index for the resulting matrix
    d_mc[index] = 0;
    for(int i = 0; i < n; i++) {
        d_mc[index] += d_ma[i + row*n]*d_mb[col+i*k];
    }
}

__global__ void valid_matrix_dot(int m, int n, int k, float *d_ma, float *d_mb, float *d_mc) {
    // Make a mega matrix of the block matrices of the device xd
    int col = threadIdx.x + blockIdx.x*blockDim.x;
    int row = threadIdx.y + blockIdx.y*blockDim.y;
    if(k > col && m > row) {
        matrix_dot(col, row, m, n, k, d_ma, d_mb, d_mc);
    }
}

void print_matrix(int m, int n, float *a) {
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            fprintf(stdout, "%f ", a[j+i*n]);
        }
        fprintf(stdout, "\n");
    }
    fprintf(stdout, "\n");
}

int main() {
    float *ma, *mb, *mc;
    float *d_ma, *d_mb, *d_mc;
    int m, n, n2, k, blocks;

    /* input first matrix */
    fprintf(stdout, "Give me the dimensions of the first matrix: ");
    fscanf(stdin, "%i %i", &m, &n);
    ma = (float *)malloc(sizeof(float)*m*n);

    fprintf(stdout, "Give me the first matrix, separating columns with space and rows with endline.\n");
    for(int i = 0; i < m; i++) {
        for(int j = 0; j < n; j++) {
            fscanf(stdin, "%f", &ma[j+i*n]);
        }
    }

    /* Input second matrix */
    fprintf(stdout, "Give me the dimensions of the second matrix: ");
    fscanf(stdin, "%i %i", &n2, &k);
    if(n != n2) {
        fprintf(stdout, "Incorrect Dimensions!\n");
        return -1;
    }
    mb = (float *)malloc(sizeof(float)*n*k);

    fprintf(stdout, "Give me the second matrix, separating columns with space and rows with endline.\n");
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < k; j++) {
            fscanf(stdin, "%f", &mb[j+i*k]);
        }
    }

    /* Make space for third matrix */
    mc = (float *)malloc(sizeof(float)*m*k);

    hipMalloc((void**)&d_ma, sizeof(float)*m*n);
    hipMalloc((void**)&d_mb, sizeof(float)*n*k);
    hipMalloc((void**)&d_mc, sizeof(float)*m*k);
    hipMemcpy(d_ma, ma, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_mb, mb, sizeof(float)*n*k, hipMemcpyHostToDevice);
    hipMemcpy(d_mc, mc, sizeof(float)*m*k, hipMemcpyHostToDevice);

    /* call function */
    blocks = ceil((THREADS_PER_BLOCK+m*k)/THREADS_PER_BLOCK);
    dim3 Blocks(blocks, blocks);
    dim3 Threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    valid_matrix_dot<<<Blocks, Threads>>>(m, n, k, d_ma, d_mb, d_mc);

    /* cpy result */
    hipMemcpy(mc, d_mc, sizeof(float)*m*k, hipMemcpyDeviceToHost);

    /* print result */
    //print_matrix(m, n, ma);
    //print_matrix(n, k, mb);
    print_matrix(m, k, mc);

    free(ma);free(mb);free(mc);
    hipFree(d_ma);hipFree(d_mb);hipFree(d_mc);
    return 0;
}
