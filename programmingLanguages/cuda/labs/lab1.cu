/*
   Pedro Luis González Roa
   A01651517
   Cuda lab 1
*/

// %%cu
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1000000000;
const int THREADS = 256;

__global__ void calc_pi(double* area){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    double mid, height, width;
    if(index < N){
        width = 1.0 / (double) N;
        mid = (index + .5) * width;
        height = 4.0/ (1.0 + mid * mid);
        area[index]= width * height;
	index += blockDim.x * gridDim.x;
    }
}

int main(void){
    double res, *area, *d_area;
    // Allocate memory for area
    area = (double*) malloc(N * sizeof(double));
    // Allocate memory for device area
    hipMalloc((void **)&d_area, N * sizeof(double));
    // Call GPU function
    calc_pi<<<(N/THREADS)+1, THREADS>>>(d_area);
    // Cpy device are to area
    hipMemcpy(area, d_area, sizeof(double) * N, hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++)
        res += area[i];

    printf("RECTANGLES: %d\n", N);
    printf("THREADS PER BLOCK: %d\n", THREADS);
    printf("RESULT: %f\n", res);

    free(area);
    hipFree(d_area);
    return 0;
}
